#include "hip/hip_runtime.h"
#include <stdio.h>
#include "GPU_Manage.h"
// #include "Types.h"
#include "Constants.h"
#include "GPU_Test.h"
#include "GPU_MBR.h"
#include "IO.h"
#include "SEQ_Overlay.h"
#include "GPU_Utility.h"
#include "Data_Visualization.h"
#include "Join.h"

// #include "GH-CUDA/ghcuda.cpp"
// #include "GH-CUDA/lib/polyclip.cpp"

hipEvent_t start_GPU, stop_GPU;

void copyPolygon(
    double **polyPX, double **polyPY, double **polyQX, double **polyQY, 
    int *bVNum, long *bVPSNum, coord_t* baseCoords, 
    int *oVNum, long *oVPSNum,coord_t* overlayCoords,
    int baseID, int overlayID) {
    int sizeP=bVNum[baseID], sizeQ=oVNum[overlayID];    
    *polyPX=(double *)malloc(sizeP*sizeof(double));
    *polyPY=(double *)malloc(sizeP*sizeof(double));
    *polyQX=(double *)malloc(sizeQ*sizeof(double));
    *polyQY=(double *)malloc(sizeQ*sizeof(double));
    printf("size %d %d %d %d\n", bVPSNum[0], bVPSNum[1], bVNum[0], bVNum[1]);
    for(int j=0, i=bVPSNum[baseID]-bVNum[baseID]; i<sizeP; j++, i+=2){
        *(*polyPX+j)=baseCoords[i];
        *(*polyPY+j)=baseCoords[i+1];
    }
    for(int j=0, i=oVPSNum[baseID]-oVNum[baseID]; i<sizeP; j++, i+=2){
        *(*polyPX+j)=overlayCoords[i];
        *(*polyPY+j)=overlayCoords[i+1];
    }
}

// int main(int argc, char* argv[]){  
int spatialJoin(int argc, char* argv[], int  **pIDList, int **qIDList, long *totalPairNum, 
    coord_t **pBaseCoords, coord_t **pOverlayCoords, 
    int **pBVNum, long **pBVPSNum, int **pOVNum, long **pOVPSNum){  

    float Join_Total_Time_SEQ=0, Join_Total_Time_GPU=0;
    hipError_t cudaMemError;
//------------------------ Console Input ---------------------------------- 
/*
First user input: dimSort
	1: Sorting just based on one dimension (default is X)
	0: Sorting based on both X and Y dimensions
Second user input: dimSelect
	If dimSort=1, this argument define which dimension should be picked for sorting (Values could be 'X' or 'Y')
*/
    int dimSort=1, dimSelect=1;
    if(argc<2){
       dimSort=1;
       dimSelect=0;
    }
    else if(argc<3){
      if(argv[1][0]=='2')dimSort=2;
      else dimSort=1;
      dimSelect=0;
    }
    else if(argc<4){
      if(argv[2][0]=='y')dimSelect=1;
      else dimSelect=0;
      if(argv[1][0]=='2'){dimSort=2;dimSelect=0;}
      else dimSort=1;
    }
//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
//=============================== SEQUENTIAL RUN ===============================
//------------------------------------------------------------------------------    
//------------------------------------------------------------------------------    
    coord_t* baseCoords=(coord_t*)malloc(MAX_POLYS_BASE*4*AVG_VERTEX_PER_BASE_POL*sizeof(coord_t));
    coord_t* overlayCoords=(coord_t*)malloc(MAX_POLYS_OVERLAY*4*AVG_VERTEX_PER_OVERLAY_POL*sizeof(coord_t));
    int *bVNum=(int*)malloc(sizeof(int)*MAX_POLYS_BASE);
    int *oVNum=(int*)malloc(sizeof(int)*MAX_POLYS_OVERLAY);
    long *bVPSNum=(long*)malloc(sizeof(long)*MAX_POLYS_BASE);
    long *oVPSNum=(long*)malloc(sizeof(long)*MAX_POLYS_OVERLAY);
    long bPolNum, oPolNum, bVNumSum=0, oVNumSum=0;    
    mbr_t* seqMBR=(mbr_t*)malloc(MAX_POLYS_BASE*4*sizeof(mbr_t));
    mbr_t* seqOMBR=(mbr_t*)malloc(MAX_POLYS_OVERLAY*4*sizeof(mbr_t));
    coord_t* seqMBR2=(coord_t*)malloc(MAX_POLYS_BASE*4*sizeof(coord_t));
    coord_t* seqOMBR2=(coord_t*)malloc(MAX_POLYS_OVERLAY*4*sizeof(coord_t));

    //=================== Reading First(base) Polygon ==========================

    char baseFileName[100], overlayFileName[100];
    switch(DATASET){
       case 1:
        //  strcpy(baseFileName, "/pylon5/cc560kp/danialll/Text_Datasets/admin_states.txt");
	    //  strcpy(overlayFileName, "/pylon5/cc560kp/danialll/Text_Datasets/urban_areas.txt");
        
        //  printf("\nDataset: admin - urban\n");
        //  strcpy(baseFileName, "../datasets/datasets/admin_states.txt");
	    //  strcpy(overlayFileName, "../datasets/datasets/urban_areas.txt");

         printf("Dataset: Ocean - Land\n");
         strcpy(baseFileName, "../datasets/datasets/ne_10m_ocean.txt");
	     strcpy(overlayFileName, "../datasets/datasets/ne_10m_land.txt");
        //  strcpy(overlayFileName, "../datasets/datasets/ne_10m_ocean.txt");
	    //  strcpy(baseFileName, "../datasets/datasets/ne_10m_land.txt");
         break;
       case 2:
         strcpy(baseFileName, "/pylon5/cc560kp/danialll/Text_Datasets/bases_242.txt");
         strcpy(overlayFileName, "/pylon5/cc560kp/danialll/Text_Datasets/overlay_300.txt");
         printf("\nDataset: bases - overlay\n");
         break;
       case 3:
         strcpy(baseFileName, "/pylon5/cc560kp/danialll/Text_Datasets/block_boundaries.txt");
         strcpy(overlayFileName, "/pylon5/cc560kp/danialll/Text_Datasets/water_bodies.txt");
         printf("\nDataset: boundaries - water\n");
         break;
       case 4:
         strcpy(baseFileName, "/pylon5/cc560kp/danialll/Text_Datasets/postal.txt");
         strcpy(overlayFileName, "/pylon5/cc560kp/danialll/Text_Datasets/sports.txt");
         printf("\nDataset: postal - sports\n");
         break;
    }
    bPolNum=ReadTextFormatPolygon2WithVector(baseFileName,bVNum, bVPSNum, seqMBR, seqMBR2, baseCoords, &bVNumSum, 1, MAX_POLYS_BASE, pPolygons);    
    printf("\n%lu Polygons with %lu vertices in total.\n",bPolNum,bVNumSum);
    oPolNum=ReadTextFormatPolygon2WithVector(overlayFileName, oVNum, oVPSNum, seqOMBR, seqOMBR2, overlayCoords, &oVNumSum, 1, MAX_POLYS_OVERLAY, qPolygons); 
    printf("\n%lu Polygons with %lu vertices in total.\n",oPolNum,oVNumSum);

    //==========================================================================
    printf("\npPolygons size %d qPolygons size %d\n", pPolygons.size(), qPolygons.size());
// PrintPolygon(baseCoords+2*bVPSNum[1485], bVNum[1486]);
PrintPolygon(baseCoords, 5);
printf("\n\n\n");
// PrintPolygon(overlayCoords+2*oVPSNum[10], oVNum[11]);
PrintPolygon(overlayCoords, 5);
//return;

printf("size %d %d %d %d %d\n", bVNum[0], bVNum[1], bVNum[2], bVPSNum[0], bVPSNum[1]);
// PrintPolygon(baseCoords+2*bVPSNum[0], bVNum[1]);

// =======================**************========================================
/* CPU data structures
----------------------------------------------------
bPolNum: total # polygons in the base layer
bVNumSum: sum of vertices in the base layer

bVNum[]: size of each polygon
bVPSNum[]: prefixsum of bVNum[]

baseCoords[]:coordinates of base polygon {x_i, y_i} pairs in the same array

oPolNum: total # polygons in the overlay layer
oVNumSum: sum of vertices in the overlay layer

oVNum[]: size of each polygon
oVPSNum[]: prefixsum of bVNum[]

overlayCoords[]:coordinates of overlay polygon {x_i, y_i} pairs in the same array
*/
// =======================**************========================================

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//================================== GPU RUN ===================================
//------------------------------------------------------------------------------    
//------------------------------------------------------------------------------    

// =======================**************========================================
/*CPU GPU data structures
----------------------------------------------------
bVNum[] dbVNum[]: size of each polygon
bVPSNum[] dbVPSNum[]: prefixsum of bVNum[]

baseCoords[] bCoords[]:coordinates of base polygon {x_i, y_i} pairs in the same array


oVNum[] doVNum[]: size of each polygon
oVPSNum[] doVPSNum[]: prefixsum of bVNum[]

overlayCoords[] oCoords[]:coordinates of overlay polygon {x_i, y_i} pairs in the same array
*/
// =======================**************========================================


//=========================== Reseting GPU device ==============================
    hipError_t error_reset=hipDeviceReset();    
    if(error_reset!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error_reset) );
       exit(-1);
    }
    hipDeviceReset();
    //size_t mem_free_0,mem_total_0;
    //hipMemGetInfo  (&mem_free_0, &mem_total_0);
    //printf("\nFree: %lu  , Total: %lu\n",mem_free_0,mem_total_0);
    
    //==================== Running Kernel (CreateMBR) =========================
    //====================== Transfering data to GPU ==========================
    StartTimer(&start_GPU, &stop_GPU);
   
    int *dbVNum, *doVNum;
    coord_t *oCoords, *bCoords;
    mbr_t *doMBR, *dbMBR;
    long *dbVPSNum, *doVPSNum;

    //----------- Transfering polygon number variables to GPU ---------------
    CopyToGPU((void**)&dbVNum, bVNum, sizeof(int)*bPolNum, "dbVNum", 1);
    CopyToGPU((void**)&doVNum, oVNum, sizeof(int)*oPolNum, "doVNum", 1);
    CopyToGPU((void**)&dbVPSNum, bVPSNum, sizeof(long)*bPolNum, "dbVPSNum", 1);
    CopyToGPU((void**)&doVPSNum, oVPSNum, sizeof(long)*oPolNum, "doVPSNum", 1);
    //-----------------------------------------------------------------------
    //------------- Transfering polygon coordinates to GPU i-----------------
    CopyToGPU((void**)&bCoords, baseCoords, sizeof(coord_t)*2*bVNumSum, "bCoords", 1);
    CopyToGPU((void**)&oCoords, overlayCoords, sizeof(coord_t)*2*oVNumSum, "oCoords", 1);
    //-----------------------------------------------------------------------
    //----------------------- Transfering MBRs to GPU -----------------------
    CopyToGPU((void**)&dbMBR, seqMBR, 4*sizeof(mbr_t)*bPolNum, "dbMBR", 1);
    CopyToGPU((void**)&doMBR, seqOMBR, 4*sizeof(mbr_t)*oPolNum, "doMBR", 1);
    //-----------------------------------------------------------------------
    GPUSync("Transfering data to GPU");

    float runningTime_GPU_TransferData;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_TransferData);
    printf("\n\nGPU running time for transfering data to GPU: %f (%f)\n",runningTime_GPU_TransferData, Join_Total_Time_GPU);
//==============================================================================

//GPUPrintVector(4*oPolNum, doMBR2, 0);
//SEQMBROverlap(bPolNum, oPolNum, seqMBR, seqOMBR, -1);
//return;


   /*

   SpatialJoin(int* bPolNum, int* oPolNum, int* bVNum, int* oVNum, coord_t* bCoord, coord_t* oCoord, coord_t * bMBR, coord_t* oMBR, mbr_t* bMBR2, mbr_t* oMBR2, int** jxyVector, int* pairNum);
  
   Input parameters:
   bPolNum: Number of polygons in layer 1
   bMBR: MBRs from polygons of coord_t type (float). Format (x1, y1, x2, y2) in a vector structure of size 4*bPolNum*sizeof(coord_t)
   bMBR2: MBRs from polygons of mbr_t type (long long). Same format as bMBR.
   bVNum: Number of vertices in each polygon. We have bPolNum polygons then bVNum has bPolNum int elements.
   bCoord: Vertices of all the polygons of layer b. Format (x1_0,y1_0, x2_0, y2_0,...., x1_2, y1_2,.....,x1_(bPolNum-1), y1_(bPolNum-1) )

   Output parameters:
   pairNum: Number of output pairs
   jxyVector: Output pairs in format (i1, j1, i2, j2,....) i1: index of polygon from b layer and j1 index of polygon from o layer.
   */


//--------------------------- Find Overlaping MBRs (novel approach) ---------------------------
    StartTimer(&start_GPU, &stop_GPU);

    int *djxyCounter, *djxyVector, polNum=bPolNum+oPolNum; 
    // int *djxyCounter, polNum=bPolNum+oPolNum; 
    cudaMemError=hipMalloc((void**)&djxyCounter,sizeof(int)*(polNum));

    long pairNum=SortBaseMBROverlap(bPolNum, oPolNum, dbMBR, doMBR, &djxyCounter, &djxyVector, dimSort, dimSelect);
   
    printf("\n\n\tPolygon pairs candidate: %ld\n", pairNum);
    float runningTime_GPU_overlap2;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_overlap2);
    printf("\nGPU Running Time For Computing MBR intersection (new approach %dD [dim:%c] ): %f (%f)\n",dimSort, 'X', runningTime_GPU_overlap2, Join_Total_Time_GPU);
    hipFree(doMBR);
    hipFree(dbMBR);
    hipFree(djxyCounter);

//------------------------------------------------------------------------------

//GPUPrintVector(2*pairNum, djxyVector, 0);
//return;



//--------------------------- CMF filter for Polygon Test operation --------------------------
    StartTimer(&start_GPU, &stop_GPU);
    int *djxy2IndexList, *djPiPIndexList, *dPiPFlag, *djoinFlag;
    // int *djxy2IndexList, *djPiPIndexList, *djoinFlag;
    char* dPiPType;
    long eiNum, pairNum3, pipNum, workLoadNum;
    coord_t *dcMBR, *dbMBR2, *doMBR2;
    CopyToGPU((void**)&doMBR2, seqOMBR2, sizeof(coord_t)*oPolNum*4, "doMBR2", 1);
    CopyToGPU((void**)&dbMBR2, seqMBR2, sizeof(coord_t)*bPolNum*4, "dbMBR2", 1);

    GetCMBR(pairNum, djxyVector, dbMBR2, doMBR2, &dcMBR, &djPiPIndexList, &dPiPFlag, &dPiPType, &djoinFlag, &pipNum);

    float runningTime_GPU_PiPCMF;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_PiPCMF);
    printf("\nGPU Running Time for CMF Filter for Point in Polygon Test: %f (%f)\n", runningTime_GPU_PiPCMF, Join_Total_Time_GPU);
//------------------------------------------------------------------------------

//--------------------------- Point in Polygon Test operation --------------------------
    StartTimer(&start_GPU, &stop_GPU);
    long wNum;

    wNum=PointInPolygonTest(bCoords, oCoords, pairNum, pipNum, djxyVector, djPiPIndexList, dPiPType, dbVPSNum, doVPSNum, dPiPFlag, djoinFlag);
    
    printf("\n\tNumber of within pairs: %ld\n", wNum);

    //PrintPairs(djxyVector, dPiPFlag, pairNum);
//GPUPrintVector(pairNum2, dEdgeIntersectCounter, 1);

    float runningTime_GPU_PiP;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_PiP);
    printf("\nGPU Running Time for Point in Polygon Test: %f (%f)\n", runningTime_GPU_PiP, Join_Total_Time_GPU);
 
    
    /*StartTimer(&start_GPU, &stop_GPU);
    float runningTime_SEQ_PiP;
    long wNum2=SEQ_PointInPolygonTest(baseCoords, overlayCoords, pairNum, pipNum, djxyVector, djPiPIndexList, dPiPType, bVPSNum, oVPSNum, dPiPFlag, djoinFlag);
    printf("\n\tNumber of within pairs (Sequential): %ld\n", wNum2);
    StopTimer(&start_GPU, &stop_GPU, &runningTime_SEQ_PiP);
    printf("\n\tSequential Running Time for Point in Polygon Test: %f\n", runningTime_SEQ_PiP);
    return;*/
//------------------------------------------------------------------------------



//--------------------------- Applying Common MBR Filtering (novel approach) ---------------------------
    StartTimer(&start_GPU, &stop_GPU);
    poly_size_t *dbEdgeList, *doEdgeList;
    long *dbEdgePSCounter, *doEdgePSCounter, *dWorkLoadPSCounter;
//GPUPrintVector(4*oPolNum, doMBR2, 0);
//return;


    CountCMF(bCoords, oCoords, pairNum, djxyVector, djoinFlag, dbVNum, doVNum, dbVPSNum, doVPSNum, dcMBR, &dbEdgePSCounter, &doEdgePSCounter, &dWorkLoadPSCounter, &djxy2IndexList, &dbEdgeList, &doEdgeList, &eiNum, &workLoadNum);

    //printf("\n\tPolygon pair candidates after Applying CMF filter: %ld\n", eiNum);
    float runningTime_GPU_CCMF;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_CCMF);
    printf("\nGPU Running Time for Counting Common MBR Filter: %f (%f)\n", runningTime_GPU_CCMF, Join_Total_Time_GPU);

    StartTimer(&start_GPU, &stop_GPU);

    ApplyCMF(bCoords, oCoords, pairNum, djxyVector, eiNum, djxy2IndexList, dbVNum, doVNum, dbVPSNum, doVPSNum, dcMBR, dbEdgePSCounter, doEdgePSCounter, dbEdgeList, doEdgeList);
    //GPUPrintVector(pairNum2*2, djxy2IndexList, 0);
    //GPUPrintVector(pairNum*2, djxyVector, 0);
    //GPUPrefixsumTest(dbEdgeCounter, dbEdgePSCounter, pairNum, 1);
    //GPUPrefixsumTest(doEdgeCounter, doEdgePSCounter, pairNum, 1);

    float runningTime_GPU_ACMF;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_ACMF);
    printf("\nGPU Running Time for Applying Common MBR Filter: %f (%f)\n", runningTime_GPU_ACMF, Join_Total_Time_GPU);
    hipFree(dcMBR);

//------------------------------------------------------------------------------

//--------------------------- Join/Overlay operations --------------------------
    StartTimer(&start_GPU, &stop_GPU);
    int* dSegmentIntersectJoinFlag;
    pairNum3=SegmentIntersectJoin(bCoords, oCoords, eiNum, djxyVector, djxy2IndexList, dbVPSNum, doVPSNum, dbEdgePSCounter, doEdgePSCounter, dbEdgeList, doEdgeList, &dSegmentIntersectJoinFlag);
    //pairNum3=SegmentIntersectJoin2(bCoords, oCoords, eiNum, djxyVector, djxy2IndexList, dbVPSNum, doVPSNum, dbEdgePSCounter, doEdgePSCounter, dWorkLoadPSCounter, workLoadNum, dbEdgeList, doEdgeList, &dSegmentIntersectJoinFlag);

    // PrintPairs(djxyVector, dPiPFlag, pairNum); // how to get pair IDs *************


    /*
    How to get the pairs in GPU
    djxyVector[]: ID pairs of the intersecting polygons from base and overlay {b_i,o_i}
    dPiPFlag: if a pair is intersecting, the flag vlues is 1. 
    pairNum: long value. Contains the total # intersecting pairs
    */

// GPUPrintVector(pairNum2, dEdgeIntersectCounter, 1);

    printf("\n\tActual number of intersected polygon pairs: %ld\n", pairNum3);
    float runningTime_GPU_CEI;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_CEI);
    printf("\nGPU Running Time for Counting Edge Intersecions: %f (%f)\n", runningTime_GPU_CEI, Join_Total_Time_GPU);

// return;

    StartTimer(&start_GPU, &stop_GPU);


    float runningTime_GPU_EI;
    Join_Total_Time_GPU+=StopTimer(&start_GPU, &stop_GPU, &runningTime_GPU_EI);
    printf("\nGPU Running Time for Computing Edge Intersections: %f (%f)\n", runningTime_GPU_EI, Join_Total_Time_GPU);
//------------------------------------------------------------------------------

    // GPUPrintVector(eiNum, dSegmentIntersectJoinFlag, 1);

    // return;
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------

//hipFree(djxyCounter);

//GCMF + GH merge sample code
// int baseID=260, overlayID=11589;
// double *polyPX, *polyPY, *polyQX, *polyQY;
// copyPolygon(&polyPX, &polyPY, &polyQX, &polyQY,
//             bVNum, bVPSNum, baseCoords, 
//             oVNum, oVPSNum, overlayCoords,
//             baseID, overlayID);

    // =========================================================================
    // Assign additional pointers to the input data arrays to be used in the GH code CPU
    *pBaseCoords=bCoords;
    *pOverlayCoords=oCoords;
    *pBVNum=dbVNum;
    *pBVPSNum=dbVPSNum;
    *pOVNum=doVNum; 
    *pOVPSNum=doVPSNum;
    // =========================================================================

*pIDList=(int *)malloc(sizeof(int)*pairNum);
*qIDList=(int *)malloc(sizeof(int)*pairNum);
CopyPairsToCPU(*pIDList, *qIDList, totalPairNum, djxyVector, dPiPFlag, pairNum); 

// hipDeviceReset(); ********** resets cuda environment
//==============================================================================
return 0;
}
